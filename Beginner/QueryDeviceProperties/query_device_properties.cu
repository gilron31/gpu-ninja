
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
  int device;
  hipGetDevice(&device);
  std::cout << "device: " << device << std::endl;
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);

  std::cout << "name: " << prop.name << std::endl;
  std::cout << "pciBusID: " << prop.pciBusID << std::endl;
  std::cout << "major: " << prop.major << std::endl;
  std::cout << "minor: " << prop.minor << std::endl;
  std::cout << "totalGlobalMem: " << prop.totalGlobalMem << std::endl;
  std::cout << "l2CacheSize: " << prop.l2CacheSize << std::endl;
  std::cout << "sharedMemPerMultiprocessor: " << prop.sharedMemPerMultiprocessor
            << std::endl;
  std::cout << "regsPerMultiprocessor: " << prop.regsPerMultiprocessor
            << std::endl;
  std::cout << "multiProcessorCount: " << prop.multiProcessorCount << std::endl;
}
