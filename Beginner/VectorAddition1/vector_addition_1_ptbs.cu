
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <vector>

template <class T>
__global__ void vector_addition_kernel(const T *a, const T *b, size_t size,
                                       T *c) {
  uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t stride = blockDim.x * gridDim.x;
  for (uint32_t i = tid; i < size; i += stride) {
    c[i] = a[i] + b[i];
  }
}

template <class T>
void vector_addition_impl(const std::vector<T> &a, const std::vector<T> &b,
                          std::vector<T> &c) {
  assert(a.size() == b.size());
  size_t size = a.size();
  assert(size == c.size());

  T *d_a;
  T *d_b;
  T *d_c;

  hipMalloc(&d_a, sizeof(T) * size);
  hipMalloc(&d_b, sizeof(T) * size);
  hipMalloc(&d_c, sizeof(T) * size);

  hipMemcpy(d_a, a.data(), sizeof(T) * size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b.data(), sizeof(T) * size, hipMemcpyHostToDevice);

  vector_addition_kernel<<<512, 10>>>(d_a, d_b, size, d_c);

  hipMemcpy(c.data(), d_c, sizeof(T) * size, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

template <class T>
void vector_addition_cpu(const std::vector<T> &a, const std::vector<T> &b,
                         std::vector<T> &c) {
  assert(a.size() == b.size());
  size_t size = a.size();
  assert(size == c.size());

  for (size_t i = 0; i < size; ++i) {
    c[i] = a[i] + b[i];
  }
}

std::ostream &operator<<(std::ostream &os, const std::vector<int> &input) {
  for (auto const &i : input) {
    os << i << " ";
  }
  return os;
}

int main() {
  constexpr size_t SIZE = 100000;
  std::vector<int> a(SIZE, 4);
  std::vector<int> b(SIZE, 5);
  std::vector<int> c(SIZE);
  std::vector<int> exp(SIZE);

  vector_addition_impl(a, b, c);
  vector_addition_cpu(a, b, exp);

  assert(c == exp);
  std::cout << "SUCCESS" << std::endl;
}