
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <vector>

template <class T>
__global__ void vector_addition_kernel(const T *a, const T *b, size_t size,
                                       T *rv) {
  uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t stride = blockDim.x * gridDim.x;
  for (uint32_t i = tid; i < size; i += stride) {
    rv[i] = a[i] + b[i];
  }
}

template <class T>
void vector_addition_impl(const std::vector<T> &a, const std::vector<T> &b,
                          std::vector<T> &rv) {
  assert(a.size() == b.size());
  size_t size = a.size();
  assert(size == rv.size());

  T *d_a;
  T *d_b;
  T *d_rv;

  hipMalloc(&d_a, sizeof(T) * size);
  hipMalloc(&d_b, sizeof(T) * size);
  hipMalloc(&d_rv, sizeof(T) * size);

  hipMemcpy(d_a, a.data(), sizeof(T) * size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b.data(), sizeof(T) * size, hipMemcpyHostToDevice);

  vector_addition_kernel<<<512, 10>>>(d_a, d_b, size, d_rv);

  hipMemcpy(rv.data(), d_rv, sizeof(T) * size, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_rv);
}

template <class T>
void vector_addition_cpu(const std::vector<T> &a, const std::vector<T> &b,
                         std::vector<T> &rv) {
  assert(a.size() == b.size());
  size_t size = a.size();
  assert(size == rv.size());

  for (size_t i = 0; i < size; ++i) {
    rv[i] = a[i] + b[i];
  }
}

std::ostream &operator<<(std::ostream &os, const std::vector<int> &input) {
  for (auto const &i : input) {
    os << i << " ";
  }
  return os;
}

int main() {
  constexpr size_t SIZE = 100000;
  std::vector<int> a(SIZE, 4);
  std::vector<int> b(SIZE, 5);
  std::vector<int> rv(SIZE);
  std::vector<int> exp(SIZE);

  vector_addition_impl(a, b, rv);
  vector_addition_cpu(a, b, exp);

  assert(rv == exp);
  std::cout << "SUCCESS" << std::endl;
}