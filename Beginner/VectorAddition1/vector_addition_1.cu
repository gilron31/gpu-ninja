
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <vector>

template <class T>
__global__ void vector_addition_kernel(const T *a, const T *b, size_t size,
                                       T *c) {
  /**
   * Your Code Here!
   *
   * 1. Use all threads in the grid.
   * 2. Make this kernel as simple as possible (optimizations will come later).
   */
}

template <class T>
void vector_addition_impl(const std::vector<T> &a, const std::vector<T> &b,
                          std::vector<T> &c) {
  /**
   * Your Code Here!
   *
   * 1. Don't forget input validation.
   * 2. Don't forget to free the allocated device memory.
   * 3. Time each of the sections:
   *    a. Device memory allocation.
   *    b. H2D copy.
   *    c. Kernel invocation.
   *    d. D2H copy.
   *    e. Device memory free.
   */
}

template <class T>
void vector_addition_cpu(const std::vector<T> &a, const std::vector<T> &b,
                         std::vector<T> &c) {
  assert(a.size() == b.size());
  size_t size = a.size();
  assert(size == c.size());

  for (size_t i = 0; i < size; ++i) {
    c[i] = a[i] + b[i];
  }
}

std::ostream &operator<<(std::ostream &os, const std::vector<int> &input) {
  for (auto const &i : input) {
    os << i << " ";
  }
  return os;
}

int main() {
  constexpr size_t SIZE = 100000;
  std::vector<int> a(SIZE, 4);
  std::vector<int> b(SIZE, 5);
  std::vector<int> c(SIZE);
  std::vector<int> exp(SIZE);

  vector_addition_impl(a, b, c);
  vector_addition_cpu(a, b, exp);

  assert(c == exp);
  std::cout << "SUCCESS" << std::endl;
}