
#include <hip/hip_runtime.h>
#include <iostream>

void using_runtime_api() {
  int device;
  hipGetDevice(&device);
  std::cout << "device: " << device << std::endl;
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);

  std::cout << "name: " << prop.name << std::endl;
  std::cout << "major: " << prop.major << std::endl;
  std::cout << "minor: " << prop.minor << std::endl;
  std::cout << "totalGlobalMem: " << prop.totalGlobalMem << std::endl;
  std::cout << "l2CacheSize: " << prop.l2CacheSize << std::endl;
  std::cout << "sharedMemPerMultiprocessor: " << prop.sharedMemPerMultiprocessor
            << std::endl;
  std::cout << "regsPerMultiprocessor: " << prop.regsPerMultiprocessor
            << std::endl;
  std::cout << "multiProcessorCount: " << prop.multiProcessorCount << std::endl;
}

void using_driver_api() {
  int device_count;
  hipGetDeviceCount(&device_count);
  std::cout << "Device count: " << device_count << std::endl;
  // TODO(gil): complete
}

int main() {
  std::cout << "Using CUDA runtime API" << std::endl;
  using_runtime_api();

  std::cout << "Using CUDA driver API: " << std::endl;
  using_driver_api();
}
