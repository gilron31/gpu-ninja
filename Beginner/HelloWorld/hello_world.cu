
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_world_kernel() { printf("Hello World from GPU\n"); }

int main() {
  std::cout << "Hello World from CPU" << std::endl;
  hello_world_kernel<<<1, 1>>>();
  hipDeviceSynchronize();
}
