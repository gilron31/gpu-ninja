// #include <iostream>
// #include <stdio.h>

// __global__ void hello_world_kernel(char *a)
// {
//     printf("hello world from GPU");
// }

// int main()
// {
//     std::cout << "hello world from CPU" << std::endl;
//     char *d_a;
//     cudaMalloc(&d_a, 100);
//     hello_world_kernel<<<1, 1>>>(d_a);
//     cudaDeviceSynchronize();
// }


#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDART_CHECK(fn)                                                          \
    do                                                                            \
    {                                                                             \
        hipError_t status = (fn);                                                \
        if (hipSuccess != (status))                                              \
        {                                                                         \
            fprintf(stderr, "CUDA Runtime Failure (line %d of file %s):\n\t"      \
                            "%s returned 0x%x (%s)\n",                            \
                    __LINE__, __FILE__, #fn, status, hipGetErrorString(status)); \
        }                                                                         \
    } while (0);

__global__ void helloCUDA(float f)
{
    for (size_t i = 0; i < 1; i++)
    {
        printf("Hello thread %d, f=%f\n", threadIdx.x, f);
    }
}

int main()
{
    printf("hello from cpu\n");
    helloCUDA<<<1, 5>>>(1.2345f);
    hipDeviceSynchronize();
    CUDART_CHECK(hipPeekAtLastError());
    return 0;
}